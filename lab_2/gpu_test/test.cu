#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

__global__ void test(float *A)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if(id == 0)
    A[0] = 233333;
  return;
}
int main(int argc, char **argv)
{
  char hostname[100];
  gethostname(hostname, 100); //Get my host name
  cout << hostname << " Hello World" << endl;
  float *A = NULL;
  float *B = NULL;
  A = (float*)malloc(sizeof(float));
  hipMalloc(&B, sizeof(float));
  test<<<1,1>>>(B);
  hipMemcpy(A, B, sizeof(float), hipMemcpyDeviceToHost);
  cout << A[0] << endl;
  free(A);
  hipFree(B);
  return 0;
}
